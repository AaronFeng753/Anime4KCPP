#include "hip/hip_runtime.h"
#include"CudaHelper.cuh"
#include"CudaInterface.hpp"

typedef unsigned char uchar;

#define MAX3(a, b, c) fmaxf(fmaxf(a,b),c)
#define MIN3(a, b, c) fminf(fminf(a,b),c)
#define UNFLOAT(n) ((n) >= 255 ? 255 : ((n) <= 0 ? 0 : uchar((n) + 0.5)))

inline __device__ static void getLightest(uchar4& mc, uchar4& a, uchar4& b, uchar4& c, float strength)
{
    mc = make_uchar4(
        mc.x + strength * (__fdividef(a.x + b.x + c.x, 3.0f) - mc.x) + 0.5f,
        mc.y + strength * (__fdividef(a.y + b.y + c.y, 3.0f) - mc.y) + 0.5f,
        mc.z + strength * (__fdividef(a.z + b.z + c.z, 3.0f) - mc.z) + 0.5f,
        mc.w + strength * (__fdividef(a.w + b.w + c.w, 3.0f) - mc.w) + 0.5f
    );
}

inline __device__ static void getAVerage(uchar4& mc, uchar4& a, uchar4& b, uchar4& c, float strength)
{
    mc = make_uchar4(
        mc.x + strength * (__fdividef(a.x + b.x + c.x, 3.0f) - mc.x) + 0.5f,
        mc.y + strength * (__fdividef(a.y + b.y + c.y, 3.0f) - mc.y) + 0.5f,
        mc.z + strength * (__fdividef(a.z + b.z + c.z, 3.0f) - mc.z) + 0.5f,
        0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x + 0.5f
    );
}

__global__ static void getGray(
    hipTextureObject_t srcImg, hipSurfaceObject_t dstImg,
    int W, int H
)
{
    const unsigned int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    if (x >= W || y >= H)
        return;

    float u = (x + 0.5f) / (float)(W);
    float v = (y + 0.5f) / (float)(H);

    float4 fmc = tex2D<float4>(srcImg, u, v);
    uchar4 mc = make_uchar4(
        fmc.x * 255.0f + 0.5f, fmc.y * 255.0f + 0.5f, fmc.z * 255.0f + 0.5f, fmc.w * 255.0f + 0.5f
    );
    mc.w = 0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x + 0.5f;
    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

__global__ static void pushColor(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    int W, int H, float strength
)
{
    const unsigned int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    if (x >= W || y >= H)
        return;

    uchar4 tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, __umul24(sizeof(mc), x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, __umul24(sizeof(mc), x), y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, __umul24(sizeof(mc), x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, __umul24(sizeof(mc), x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, __umul24(sizeof(mc), x), y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, __umul24(sizeof(mc), x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, __umul24(sizeof(mc), x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, __umul24(sizeof(mc), x), y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, __umul24(sizeof(mc), x + 1), y + 1, hipBoundaryModeZero);

    uchar maxD, minL;

    //top and bottom
    maxD = MAX3(bl.w, bc.w, br.w);
    minL = MIN3(tl.w, tc.w, tr.w);
    if (minL > mc.w && mc.w > maxD)
        getLightest(mc, tl, tc, tr, strength);
    else
    {
        maxD = MAX3(tl.w, tc.w, tr.w);
        minL = MIN3(bl.w, bc.w, br.w);
        if (minL > mc.w && mc.w > maxD)
            getLightest(mc, bl, bc, br, strength);
    }

    //sundiagonal
    maxD = MAX3(ml.w, mc.w, bc.w);
    minL = MIN3(tc.w, tr.w, mr.w);
    if (minL > maxD)
        getLightest(mc, tc, tr, mr, strength);
    else
    {
        maxD = MAX3(tc.w, mc.w, mr.w);
        minL = MIN3(ml.w, bl.w, bc.w);
        if (minL > maxD)
            getLightest(mc, ml, bl, bc, strength);
    }

    //left and right
    maxD = MAX3(tl.w, ml.w, bl.w);
    minL = MIN3(tr.w, mr.w, br.w);
    if (minL > mc.w && mc.w > maxD)
        getLightest(mc, tr, mr, br, strength);
    else
    {
        maxD = MAX3(tr.w, mr.w, br.w);
        minL = MIN3(tl.w, ml.w, bl.w);
        if (minL > mc.w && mc.w > maxD)
            getLightest(mc, tl, ml, bl, strength);
    }

    //diagonal
    maxD = MAX3(tc.w, mc.w, ml.w);
    minL = MIN3(mr.w, br.w, bc.w);
    if (minL > maxD)
        getLightest(mc, mr, br, bc, strength);
    else
    {
        maxD = MAX3(bc.w, mc.w, mr.w);
        minL = MIN3(ml.w, tl.w, tc.w);
        if (minL > maxD)
            getLightest(mc, ml, tl, tc, strength);
    }

    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

__global__ static void getGradient(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    int W, int H
)
{
    const unsigned int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    if (x >= W || y >= H)
        return;

    uchar4 tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, __umul24(sizeof(mc), x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, __umul24(sizeof(mc), x), y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, __umul24(sizeof(mc), x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, __umul24(sizeof(mc), x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, __umul24(sizeof(mc), x), y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, __umul24(sizeof(mc), x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, __umul24(sizeof(mc), x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, __umul24(sizeof(mc), x), y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, __umul24(sizeof(mc), x + 1), y + 1, hipBoundaryModeZero);

    const float gradX = tr.w + mr.w + mr.w + br.w - tl.w - ml.w - ml.w - bl.w;
    const float gradY = tl.w + tc.w + tc.w + tr.w - bl.w - bc.w - bc.w - br.w;

    const int grad = sqrtf(gradX * gradX + gradY * gradY);
    mc.w = (uchar)255 - UNFLOAT(grad);

    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

__global__ static void pushGradient(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    int W, int H, float strength
)
{
    const unsigned int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    const unsigned int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    if (x >= W || y >= H)
        return;

    uchar4 tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, __umul24(sizeof(mc), x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, __umul24(sizeof(mc), x), y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, __umul24(sizeof(mc), x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, __umul24(sizeof(mc), x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, __umul24(sizeof(mc), x), y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, __umul24(sizeof(mc), x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, __umul24(sizeof(mc), x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, __umul24(sizeof(mc), x), y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, __umul24(sizeof(mc), x + 1), y + 1, hipBoundaryModeZero);

    uchar maxD, minL;

    //top and bottom
    maxD = MAX3(bl.w, bc.w, br.w);
    minL = MIN3(tl.w, tc.w, tr.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage(mc, tl, tc, tr, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tl.w, tc.w, tr.w);
    minL = MIN3(bl.w, bc.w, br.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage(mc, bl, bc, br, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //sundiagonal
    maxD = MAX3(ml.w, mc.w, bc.w);
    minL = MIN3(tc.w, tr.w, mr.w);
    if (minL > maxD)
    {
        getAVerage(mc, tc, tr, mr, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tc.w, mc.w, mr.w);
    minL = MIN3(ml.w, bl.w, bc.w);
    if (minL > maxD)
    {
        getAVerage(mc, ml, bl, bc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //left and right
    maxD = MAX3(tl.w, ml.w, bl.w);
    minL = MIN3(tr.w, mr.w, br.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage(mc, tr, mr, br, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tr.w, mr.w, br.w);
    minL = MIN3(tl.w, ml.w, bl.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage(mc, tl, ml, bl, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //diagonal
    maxD = MAX3(tc.w, mc.w, ml.w);
    minL = MIN3(mr.w, br.w, bc.w);
    if (minL > maxD)
    {
        getAVerage(mc, mr, br, bc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }
    maxD = MAX3(bc.w, mc.w, mr.w);
    minL = MIN3(ml.w, tl.w, tc.w);
    if (minL > maxD)
    {
        getAVerage(mc, ml, tl, tc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    mc.w = 0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x + 0.5f;
    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

void cuRunKernelAnime4K09(const unsigned char* inputData, unsigned char* outputData, ACCudaParamAnime4K09 * param)
{
    hipError_t err = hipSuccess;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();

    hipArray_t cuArray0;
    err = hipMallocArray(&cuArray0, &channelDesc, param->orgW, param->orgH);
    CheckCudaErr(err);

    hipArray_t cuArray1;
    err = hipMallocArray(&cuArray1, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    hipArray_t cuArray2;
    err = hipMallocArray(&cuArray2, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    hipArray_t cuArray3;
    err = hipMallocArray(&cuArray3, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    struct hipResourceDesc resDesc;
    struct hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    memset(&texDesc, 0, sizeof(texDesc));

    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    resDesc.resType = hipResourceTypeArray;

    resDesc.res.array.array = cuArray0;
    hipTextureObject_t tex = 0;
    err = hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray1;
    hipSurfaceObject_t surf1 = 0;
    err = hipCreateSurfaceObject(&surf1, &resDesc);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray2;
    hipSurfaceObject_t surf2 = 0;
    err = hipCreateSurfaceObject(&surf2, &resDesc);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray3;
    hipSurfaceObject_t surf3 = 0;
    err = hipCreateSurfaceObject(&surf3, &resDesc);
    CheckCudaErr(err);

    err = hipMemcpy2DToArray(cuArray0, 0, 0, inputData,
        sizeof(uchar4) * param->orgW, sizeof(uchar4) * param->orgW, param->orgH,
        hipMemcpyHostToDevice);
    CheckCudaErr(err);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(
        (param->W + dimBlock.x - 1) / dimBlock.x,
        (param->H + dimBlock.y - 1) / dimBlock.y
    );

    {
        int i;
        getGray <<<dimGrid, dimBlock>>> (tex, surf1, param->W, param->H);
        for (i = 0; i < param->passes && i < param->pushColorCount; i++)
        {
            pushColor <<<dimGrid, dimBlock>>> (surf1, surf2, param->W, param->H, param->strengthColor);
            getGradient <<<dimGrid, dimBlock>>> (surf2, surf3, param->W, param->H);
            pushGradient <<<dimGrid, dimBlock>>> (surf3, surf1, param->W, param->H, param->strengthGradient);
        }
        while (i++ < param->passes)
        {
            getGradient <<<dimGrid, dimBlock>>> (surf1, surf2, param->W, param->H);
            pushGradient <<<dimGrid, dimBlock>>> (surf2, surf1, param->W, param->H, param->strengthGradient);
        }
    }

    err = hipMemcpy2DFromArray(outputData, sizeof(uchar4) * param->W, cuArray1, 0, 0,
        sizeof(uchar4) * param->W, param->H,
        hipMemcpyDeviceToHost);
    CheckCudaErr(err);

    hipDestroyTextureObject(tex);
    hipDestroySurfaceObject(surf1);
    hipDestroySurfaceObject(surf2);
    hipDestroySurfaceObject(surf3);

    hipFreeArray(cuArray0);
    hipFreeArray(cuArray1);
    hipFreeArray(cuArray2);
    hipFreeArray(cuArray3);
}
