#include"CudaHelper.cuh"
#include"CudaInterface.hpp"

void initCuda(const unsigned int id)
{
    hipSetDevice(id);
}

void releaseCuda()
{
    hipError_t err = hipDeviceReset();
    CheckCudaErr(err);
}
